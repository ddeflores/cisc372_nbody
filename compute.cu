#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

__global__ void computeAccelKernel(double* d_mass, vector3* d_hPos, vector3* d_accels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < NUMENTITIES && j < NUMENTITIES) {
        if (i != j) {
            vector3 distance;
            for (int k = 0; k < 3; k++) {
                distance[k] = d_hPos[i][k] - d_hPos[j][k];
            }
            double magnitude_sq = distance[0]*distance[0] + distance[1]*distance[1] + distance[2]*distance[2];
            double magnitude = sqrt(magnitude_sq);
            double force = (-GRAV_CONSTANT * d_mass[j]) / magnitude_sq;
            for (int k = 0; k < 3; k++) {
                d_accels[i * NUMENTITIES + j][k] = force * distance[k] / magnitude;
            }
        } else {
            for (int k = 0; k < 3; k++) {
                d_accels[i * NUMENTITIES + j][k] = 0.0;
            }
        }
    }
}

__global__ void updatePosVelKernel(vector3* d_hPos, vector3* d_hVel, vector3* d_accels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUMENTITIES) {
        vector3 totalAccel = {0.0, 0.0, 0.0};
        for (int j = 0; j < NUMENTITIES; j++) {
            for (int k = 0; k < 3; k++) {
                totalAccel[k] += d_accels[idx * NUMENTITIES + j][k];
            }
        }
        for (int k = 0; k < 3; k++) {
            d_hVel[idx][k] += totalAccel[k] * INTERVAL;
            d_hPos[idx][k] += d_hVel[idx][k] * INTERVAL;
        }
    }
}

void compute() {
    // Allocate memory on GPU and transfer data from CPU to GPU
    vector3 *d_hPos, *d_hVel, *d_accels;
    double* d_mass;

    hipMalloc(&d_hPos, NUMENTITIES * sizeof(vector3));
    hipMalloc(&d_hVel, NUMENTITIES * sizeof(vector3));
    hipMalloc(&d_accels,NUMENTITIES * NUMENTITIES * sizeof(vector3));
    hipMalloc(&d_mass, NUMENTITIES * sizeof(double));

    hipMemcpy(d_hPos, hPos, NUMENTITIES * sizeof(vector3), hipMemcpyHostToDevice);
    hipMemcpy(d_hVel, hVel, NUMENTITIES * sizeof(vector3), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass, NUMENTITIES  * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 16);
    dim3 dimGrid((NUMENTITIES + dimBlock.x - 1) / dimBlock.x, (NUMENTITIES + dimBlock.y - 1) / dimBlock.y);
    computeAccelKernel<<<dimGrid, dimBlock>>>(d_mass, d_hPos, d_accels);
    hipDeviceSynchronize();
    
    dim3 dimBlockSingle(256);
    dim3 dimGridSingle((NUMENTITIES + dimBlockSingle.x - 1) / dimBlockSingle.x);
    updatePosVelKernel<<<dimBlockSingle, dimGridSingle>>>(d_hPos, d_hVel, d_accels);
    hipDeviceSynchronize();

    // Transfer updated data back to CPU and free GPU memory
    hipMemcpy(hPos, d_hPos, NUMENTITIES * sizeof(vector3), hipMemcpyDeviceToHost);
    hipMemcpy(hVel, d_hVel, NUMENTITIES * sizeof(vector3), hipMemcpyDeviceToHost);
    
    hipFree(d_hPos);
    hipFree(d_hVel);
    hipFree(d_mass);
    hipFree(d_accels);
}
